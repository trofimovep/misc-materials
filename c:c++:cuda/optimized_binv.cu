#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>
#include <vector>
#include <memory>
#include <stdexcept>
#include <cmath>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>


// use this libs (and methods like hipsolverDnDgesvd) to optimize the code 
